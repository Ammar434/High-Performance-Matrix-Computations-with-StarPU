#include "hip/hip_runtime.h"
#include "headers.hpp"

#ifdef USE_CUDA
template void cuda_mult<float>(void *buffers[], void *_args);
template void cuda_mult<double>(void *buffers[], void *_args);
template void fill_value_matrix_cuda<float>(void *buffers[], void *_args);
template void fill_value_matrix_cuda<double>(void *buffers[], void *_args);

template void assert_equal_gpu<float>(void* buffers[], void* cl_args);
template void assert_equal_gpu<double>(void* buffers[], void* cl_args);

#endif
template <typename DataType>
void cuda_mult(void *buffers[], void *_args)
{
  DataType alpha, beta;
  DataType *subA;
  DataType *subB;
  DataType *subC;
  char transA, transB;
  int nxC, nyC, nyA;
  int ldA, ldB, ldC;
  starpu_codelet_unpack_args(_args, &alpha, &transA, &transB, &beta);
  beta = 1.0;
  subA = (DataType *)(STARPU_MATRIX_GET_PTR(buffers[0]));
  subB = (DataType *)(STARPU_MATRIX_GET_PTR(buffers[1]));
  subC = (DataType *)(STARPU_MATRIX_GET_PTR(buffers[2]));

  nxC = static_cast<int>(STARPU_MATRIX_GET_NX(buffers[2]));
  nyC = static_cast<int>(STARPU_MATRIX_GET_NY(buffers[2]));
  nyA = static_cast<int>(STARPU_MATRIX_GET_NY(buffers[0]));

  ldA = static_cast<int>(STARPU_MATRIX_GET_LD(buffers[0]));
  ldB = static_cast<int>(STARPU_MATRIX_GET_LD(buffers[1]));
  ldC = static_cast<int>(STARPU_MATRIX_GET_LD(buffers[2]));
  //std::cout << "\n"
  //          << nxC << " " << nyC << " " << nyA << "\n";
  //hipStream_t stream;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  //hipStreamCreate(&stream);
  //hipblasSetStream(handle,stream);
  cublas<DataType>::gemm(
      handle,
      (transA == 'N' || transA == 'n') ? HIPBLAS_OP_N : HIPBLAS_OP_T,
      (transB == 'N' || transB == 'n') ? HIPBLAS_OP_N : HIPBLAS_OP_T,
      nxC, nyC, nyA,
      alpha,
      subA, ldA,
      subB, ldB,
      beta,
      subC, ldC);
  hipblasDestroy(handle);
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}

// static __global__ void cuda_mult_kernel(uint32_t nxC, uint32_t nyC, uint32_t nyA,
// 										uint32_t ldA, uint32_t ldB, uint32_t ldC,
// 										float *subA, float *subB, float *subC)
// {
// 	uint32_t id, i, j, k;
// 	float sum;
// 	id = blockIdx.x * blockDim.x + threadIdx.x;
// 	i = id % nxC;
// 	j = id / nxC;
// 	if (j >= nyC)
// 	{
// 		return;
// 	}
// 	sum = 0.;
// 	for (k = 0; k < nyA; k++)
// 	{
// 		sum += subA[i + k * ldA] * subB[k + j * ldB];
// 	}
// 	subC[i + j * ldC] = sum;
// }

// extern "C" void cuda_mult(void *descr[], void *arg)
// {
// 	(void)arg;
// 	float *d_subA, *d_subB, *d_subC;
// 	uint32_t nxC, nyC, nyA;
// 	uint32_t ldA, ldB, ldC;
// 	uint32_t nblocks;

// 	/* ptr gives a pointer to the first element of the local copy */
// 	d_subA = (float *)STARPU_MATRIX_GET_PTR(descr[0]);
// 	d_subB = (float *)STARPU_MATRIX_GET_PTR(descr[1]);
// 	d_subC = (float *)STARPU_MATRIX_GET_PTR(descr[2]);

// 	nxC = STARPU_MATRIX_GET_NX(descr[2]);
// 	nyC = STARPU_MATRIX_GET_NY(descr[2]);
// 	nyA = STARPU_MATRIX_GET_NY(descr[0]);

// 	ldA = STARPU_MATRIX_GET_LD(descr[0]);
// 	ldB = STARPU_MATRIX_GET_LD(descr[1]);
// 	ldC = STARPU_MATRIX_GET_LD(descr[2]);

// 	nblocks = (nxC * nyC + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
// 	cuda_mult<<<nblocks, THREADS_PER_BLOCK, 0, starpu_cuda_get_local_stream()>>>(nxC, nyC, nyA, ldA, ldB, ldC, d_subA, d_subB, d_subC);

// 	hipError_t status = hipGetLastError();
// 	if (status != hipSuccess)
// 		STARPU_CUDA_REPORT_ERROR(status);
// }
template <typename DataType>
void fill_value_matrix_cuda(void* buffers[], void* cl_args)
{
  DataType* A;
  DataType value;
  int nxA, nyA, ldA;
  // unpack arguments
  starpu_codelet_unpack_args(cl_args, &value);

  // Matrix info
  A = (DataType *)(STARPU_MATRIX_GET_PTR(buffers[0]));
  nxA = static_cast<int>(STARPU_MATRIX_GET_NX(buffers[0]));
  nyA = static_cast<int>(STARPU_MATRIX_GET_NY(buffers[0]));
  ldA = static_cast<int>(STARPU_MATRIX_GET_LD(buffers[0]));
  // Cuda stream and Cublas  handle
  //hipStream_t stream;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  //hipStreamCreate(&stream);
  //hipblasSetStream(handle,stream);
  // Fill matrix using cuda
  cuextensions<DataType>::fill(value, nxA, nyA, A,
                		  ldA, starpu_cuda_get_local_stream());

  // synchronize stream
  hipStreamSynchronize(starpu_cuda_get_local_stream());

  // destroy resource
  hipblasDestroy(handle);
  //hipStreamDestroy(stream);

}


template <typename DataType>
void assert_equal_gpu(void* buffers[], void* cl_args)
{
    DataType* A, *B;
    int nxA, nyA, ldA, ldB;
    A = (DataType *)(STARPU_MATRIX_GET_PTR(buffers[0]));
    nxA = static_cast<int>(STARPU_MATRIX_GET_NX(buffers[0]));
    nyA = static_cast<int>(STARPU_MATRIX_GET_NY(buffers[0]));
    ldA = static_cast<int>(STARPU_MATRIX_GET_LD(buffers[0]));
    B = (DataType *)(STARPU_MATRIX_GET_PTR(buffers[1]));
    ldB = static_cast<int>(STARPU_MATRIX_GET_LD(buffers[1]));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    bool is_equal = cuextensions<DataType>::test_equals(nxA, nyA, A, ldA,
                                         B,ldB, starpu_cuda_get_local_stream());
    // synchronize stream
    hipStreamSynchronize(starpu_cuda_get_local_stream());

    // destroy resource
    hipblasDestroy(handle);
 
    if(is_equal)std::cout << "Equal\n";
    else std::cout << "Not Equal \n";
}
